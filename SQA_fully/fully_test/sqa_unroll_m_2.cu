#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hip/hip_runtime_api.h"

#define N 16384
#define THREADS 64
#define TIMES 1

#define MAX 4294967295.0
#define STEP 100
#define M 64

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert (hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ uint xorshift32 (uint *state)
{
    uint x = *state;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    *state = x;
    return x;
}

__global__ void preapare_spins (char *spins, char *spins_out, uint *randvals, uint *randvals2)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    uint rand = randvals[idx];

    // intializing spins
    #pragma unroll
    for (int m = 0; m < M; m++) {
        char temp = (char)(((xorshift32(&rand) & 1) << 1) - 1);
        spins[idx*M+m] = temp;
        spins_out[idx*M+m] = temp;
	}
    randvals[idx] = rand;
	randvals2[idx] = rand;
}

__global__ void preapare_sigmas (char *spins, int *sigmas, int *sigmas_out, char *couplings)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // intializing sigmas
    for (int m = 0; m < M; m++) {
        sigmas[idx*M+m] = 0;
        sigmas_out[idx*M+m] = 0;
    }

    #pragma unroll 16
    for (int i = 0; i < N; i++) {
        #pragma unroll
        for (int m = 0; m < M; m++) {
            sigmas[idx*M+m] += spins[i*M+m]*couplings[idx*N+i];
        }
        #pragma unroll
		for (int m = 0; m < M; m++) {
		    sigmas_out[idx*M+m] = sigmas[idx*M+m];
		}
    }
}

__global__ void update_sigmas (int iter, 
                               char *spins, 
                               int *sigmas, 
                               char *couplings_buf,
                               float J_perp,
                               float beta,
                               uint *randvals,
							   char *spins_out,
							   int *sigmas_out,
							   uint *randvals_out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int m = 0; m < M; m++) { 
    	int target_spin = iter - m;
        int target_pos = target_spin*M+m;
        int which_layer = m;
        int which_spin = i;
		int idx = i*M+m;

		if (target_spin >= 0 && target_spin < N) {
	        uint rand = randvals[target_spin];
			int upper = (which_layer == 0 ? M-1 : which_layer-1);
			int lower = (which_layer == M-1 ? 0 : which_layer+1);
			float delta = 2.*(float)M*(float)spins[target_pos]*
					((float)sigmas[target_pos] - (float)M*J_perp*(float)(spins[target_spin*M+upper] + spins[target_spin*M+lower]));
			if ( (-log((float)rand / (float) MAX) / beta) > delta ) {
				sigmas_out[idx] = sigmas[idx] - 2*spins[target_pos]*couplings_buf[target_spin*N+which_spin];
				if (idx == target_pos) {  
					spins_out[idx] = -spins[idx];
				} else {
					spins_out[idx] = spins[idx];
				}
			} else {
				sigmas_out[idx] = sigmas[idx];
				spins_out[idx] = spins[idx];
			}
	        randvals_out[target_spin] = rand;
		}
    }
}

void usage () 
{
    printf("Usage:\n");
    printf("       ./sqa [spin configuration]\n");
    exit(0);
}

int main (int argc, char *argv[]) 
{
    if (argc != 2) 
        usage();

    // initialize couplings
    char *couplings, *couplings_buf;
    couplings = (char*)malloc(N*N*sizeof(char));
    memset(couplings, '\0', N*N*sizeof(char));
    gpuErrchk( hipMalloc(&couplings_buf, N*N*sizeof(char)) );

    // Read couplings file 
    FILE *instance = fopen(argv[1], "r");
    assert(instance != NULL);
    int a, b, w;
    fscanf(instance, "%d", &a);
    while (!feof(instance)) {
        fscanf(instance, "%d%d%d", &a, &b, &w);
        assert(a != b); // not dealing with external field
        couplings[a * N + b] = w;
        couplings[b * N + a] = w;
    }
    fclose(instance);

    // copy couplings to target device
    gpuErrchk( hipMemcpy(couplings_buf, couplings, N*N*sizeof(char), hipMemcpyHostToDevice) );

    // random number generation
    uint *randvals, *initRand;
    gpuErrchk( hipMalloc(&randvals, N * sizeof(uint)) );
    initRand = (uint*)malloc(N*sizeof(uint));
    for (int i = 0; i < N; i++)
        initRand[i] = i;
    gpuErrchk( hipMemcpy(randvals, initRand, N*sizeof(uint), hipMemcpyHostToDevice) );
    uint *randvals2;
    gpuErrchk( hipMalloc(&randvals2, N * sizeof(uint)) );

    // initialize spins
    char *s_buf, *s;
    s = (char*)malloc(M*N*sizeof(char));
    gpuErrchk( hipMalloc(&s_buf, M*N*sizeof(char)) );
    char *s_out_buf;
    gpuErrchk( hipMalloc(&s_out_buf, M*N*sizeof(char)) );
    
    // initialize 
    int *sigma_buf;
    gpuErrchk( hipMalloc(&sigma_buf, M*N*sizeof(int)) );
    int *sigma_out_buf;
    gpuErrchk( hipMalloc(&sigma_out_buf, M*N*sizeof(int)) );

    // launching kernel
    dim3 grid(N/THREADS), block(THREADS);
    int results[TIMES] = {0};
    float increase = (8 - 1/(float)16) / (float)STEP;
    float G0 = 8.;
    
    for (int t = 0; t < TIMES; t++) {
        float beta = 1/(float)16;
        
        // initialize spins and sigmas
        preapare_spins<<<grid, block>>>(s_buf, s_out_buf, randvals, randvals2);
        preapare_sigmas<<<grid, block>>>(s_buf, sigma_buf, sigma_out_buf, couplings_buf);

        double curr = 0.;

        for (int p = 0; p < STEP; p++) {
            float Gamma = G0*(1.-(float)p/(float)STEP);
            float J_perp = -0.5*log(tanh((Gamma/M)*beta))/beta;
            clock_t begin = clock();
            for (int a = 0; a < M+N-1; a++) {
				if (a % 2 == 0) {
					update_sigmas<<<grid, block>>>(a, s_buf, sigma_buf,
									   couplings_buf, J_perp, beta, randvals, s_out_buf, sigma_out_buf, randvals2);
				} else {
					update_sigmas<<<grid, block>>>(a, s_out_buf, sigma_out_buf,
									   couplings_buf, J_perp, beta, randvals2, s_buf, sigma_buf, randvals);
				}
            }
            beta += increase;
            clock_t end = clock();
            double duration = (double)(end-begin) / CLOCKS_PER_SEC;
            curr += duration;

            gpuErrchk( hipMemcpy(s, s_buf, M*N*sizeof(char), hipMemcpyDeviceToHost) );
            int E = 0;
            for (int i = 0; i < N; i++)
                for (int j = i+1; j < N; j++)
                    E += -s[i*M+0]*s[j*M+0]*couplings[i*N+j];
            results[t] = E;
            // printf("curr: %10lf, energy: %10d\n", curr, E);
        }
        printf("Per time step: %10lf, M: %10d, N: %10d\n", curr/(float)STEP, M, N);
        
    }

    // Write statistics to file
    FILE *output;
    output = fopen("output.txt", "w");
    for (int i = 0; i < TIMES; i++)
         fprintf(output, "%d\n", results[i]);
    fclose(output);

    // Release Objects
    free(couplings);
    free(initRand);
    free(s);
    hipFree(couplings_buf);
    hipFree(s_buf);
    hipFree(sigma_buf);
    hipFree(randvals);
    return 0;
}
